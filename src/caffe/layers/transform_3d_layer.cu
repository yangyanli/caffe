#include "hip/hip_runtime.h"
#include "caffe/util/field_operations.hpp"

#include "caffe/layers/transform_3d_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Transform3DForward(const int num_grids, const int grid_dim, const int batch_size, const int num_transformations,
    const Dtype pad_value, const Dtype* bottom_data, const Dtype* transformations, Dtype* top_data,
    const int len_transformation_param, const int field_channels) {
  const int t_grid_idx = blockDim.x*blockIdx.x + threadIdx.x;
  // One thread for each grid
  if(t_grid_idx < num_grids) {
    Dtype c_offset = (grid_dim-1)/2.0;
    int grid_dim_1 = grid_dim-1;
    const int yz = grid_dim*grid_dim;
    for (int b_batch_idx = 0; b_batch_idx < batch_size; ++ b_batch_idx) {
      int offset = b_batch_idx * num_transformations;
      for(int transformation_idx = 0; transformation_idx < num_transformations; ++ transformation_idx) {
        int t_batch_idx = offset + transformation_idx;

        int z = t_grid_idx%grid_dim;
        int y = (t_grid_idx/grid_dim)%grid_dim;
        int x = t_grid_idx/yz;

        Dtype xx = x+0.5-c_offset;
        Dtype yy = y+0.5-c_offset;
        Dtype zz = z+0.5-c_offset;

        const Dtype* t = transformations+t_batch_idx*len_transformation_param;
        Dtype bx = t[0]*xx + t[1]*yy + t[2]*zz + t[3] + c_offset - 0.5;
        Dtype by = t[4]*xx + t[5]*yy + t[6]*zz + t[7] + c_offset - 0.5;
        Dtype bz = t[8]*xx + t[9]*yy + t[10]*zz + t[11] + c_offset - 0.5;

        Dtype* t_data = top_data + (t_batch_idx*num_grids+t_grid_idx)*field_channels;
        if(bx >= 0 && bx < grid_dim
            && by >= 0 && by < grid_dim
            && bz >= 0 && bz < grid_dim) {
          int x0, y0, z0, x1, y1, z1;
          SnapGrid_gpu(bx, x0, x1, grid_dim_1);
          SnapGrid_gpu(by, y0, y1, grid_dim_1);
          SnapGrid_gpu(bz, z0, z1, grid_dim_1);
          Dtype x_x0 = bx-x0;
          Dtype y_y0 = by-y0;
          Dtype z_z0 = bz-z0;
          Dtype x1_x = x1-bx;
          Dtype y1_y = y1-by;
          Dtype z1_z = z1-bz;
          Interpolate_gpu(bottom_data, b_batch_idx, x0, y0, z0, x1, y1, z1,
            x_x0, y_y0, z_z0, x1_x, y1_y, z1_z, grid_dim, grid_dim, grid_dim,
            t_data, field_channels);
        } else {
          for (int i = 0; i < field_channels; ++ i) {
            t_data[i] = pad_value;
          }
        }
      } /* transformation_idx */
    } /* b_batch_idx */
  }
}

template <typename Dtype>
void Transform3DLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int field_num = bottom.size()-1;
  ForwardLabel(bottom[field_num+1], top[field_num+1]);
  if (output_inverse_transformations_) {
    ForwardInverseTransformations(&transformations_, top[field_num+2]);
  }

  for (int i = 0; i < field_num; ++ i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const vector<int>& field_shape = bottom[i]->shape();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int batch_size = field_shape[0];
    const int grid_dim = field_shape[1];
    const int num_grids = grid_dim*grid_dim*grid_dim;
    int field_channels = (field_shape.size() == 5)?(field_shape.back()):(1);
    const Dtype* transformations_data = transformations_.gpu_data();
  
    // NOLINT_NEXT_LINE(whitespace/operators)
    Transform3DForward<Dtype><<<CAFFE_GET_BLOCKS(num_grids), CAFFE_CUDA_NUM_THREADS>>>(num_grids, grid_dim, batch_size, num_transformations_,
        pad_value_, bottom_data, transformations_data, top_data, len_transformation_param, field_channels);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(Transform3DLayer);

}  // namespace caffe
