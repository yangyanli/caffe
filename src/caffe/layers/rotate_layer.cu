#include "hip/hip_runtime.h"
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layers/rotate_layer.hpp"

namespace caffe {

template<typename Dtype>
__device__ void SnapGrid_gpu(Dtype& value, int& value_0, int& value_1, const int max) {
  if (value > 0 && value < max) {
    value_0 = floor(value);
  } else if (value <= 0) {
    value = 0;
    value_0 = 0;
  } else /*(value >= max)*/ {
    value = max;
    value_0 = max-1;
  }
  value_1 = value_0 + 1;
}

template<typename Dtype>
__device__ Dtype Interpolate_gpu(const Dtype* df, const int batch_idx,
  const int x0, const int y0, const int z0,
  const int x1, const int y1, const int z1,
  const Dtype x_x0, const Dtype y_y0, const Dtype z_z0,
  const Dtype x1_x, const Dtype y1_y, const Dtype z1_z,
  const int df_dim_x, const int df_dim_y, const int df_dim_z) {
  int b_offset_000 = ((batch_idx * df_dim_x + x0) * df_dim_y + y0) * df_dim_z + z0;
  int b_offset_001 = ((batch_idx * df_dim_x + x0) * df_dim_y + y0) * df_dim_z + z1;
  int b_offset_010 = ((batch_idx * df_dim_x + x0) * df_dim_y + y1) * df_dim_z + z0;
  int b_offset_011 = ((batch_idx * df_dim_x + x0) * df_dim_y + y1) * df_dim_z + z1;
  int b_offset_100 = ((batch_idx * df_dim_x + x1) * df_dim_y + y0) * df_dim_z + z0;
  int b_offset_101 = ((batch_idx * df_dim_x + x1) * df_dim_y + y0) * df_dim_z + z1;
  int b_offset_110 = ((batch_idx * df_dim_x + x1) * df_dim_y + y1) * df_dim_z + z0;
  int b_offset_111 = ((batch_idx * df_dim_x + x1) * df_dim_y + y1) * df_dim_z + z1;

  Dtype v000 = df[b_offset_000];
  Dtype v001 = df[b_offset_001];
  Dtype v010 = df[b_offset_010];
  Dtype v011 = df[b_offset_011];
  Dtype v100 = df[b_offset_100];
  Dtype v101 = df[b_offset_101];
  Dtype v110 = df[b_offset_110];
  Dtype v111 = df[b_offset_111];

  Dtype c00 = v000*x1_x+v100*x_x0;
  Dtype c10 = v010*x1_x+v110*x_x0;
  Dtype c01 = v001*x1_x+v101*x_x0;
  Dtype c11 = v011*x1_x+v111*x_x0;

  Dtype c0 = c00*y1_y+c10*y_y0;
  Dtype c1 = c01*y1_y+c11*y_y0;

  return c0*z1_z+c1*z_z0;
}


template <typename Dtype>
__global__ void RotateForward(const int num_grids, const int grid_dim, const int batch_size, const int num_rotation,
    const Dtype pad_value, const Dtype* bottom_data, const Dtype* rotations, Dtype* top_data) {
  const int t_grid_idx = blockDim.x*blockIdx.x + threadIdx.x;
  // One thread for each grid
  if(t_grid_idx < num_grids) {
    Dtype c_offset = (grid_dim-1)/2.0;
    int grid_dim_1 = grid_dim-1;
    const int yz = grid_dim*grid_dim;
    for (int b_batch_idx = 0; b_batch_idx < batch_size; ++ b_batch_idx) {
      int offset = b_batch_idx * num_rotation;
      for(int rotation_idx = 0; rotation_idx < num_rotation; ++ rotation_idx) {
        int t_batch_idx = offset + rotation_idx;

        int r_offset = t_batch_idx*9;
        Dtype r00 = rotations[r_offset++];
        Dtype r01 = rotations[r_offset++];
        Dtype r02 = rotations[r_offset++];
        Dtype r10 = rotations[r_offset++];
        Dtype r11 = rotations[r_offset++];
        Dtype r12 = rotations[r_offset++];
        Dtype r20 = rotations[r_offset++];
        Dtype r21 = rotations[r_offset++];
        Dtype r22 = rotations[r_offset++];

        int tz = t_grid_idx%grid_dim;
        int ty = (t_grid_idx/grid_dim)%grid_dim;
        int tx = t_grid_idx/yz;

        Dtype txx = tx+0.5-c_offset;
        Dtype tyy = ty+0.5-c_offset;
        Dtype tzz = tz+0.5-c_offset;

        Dtype bx = r00*txx + r01*tyy + r02*tzz + c_offset;
        Dtype by = r10*txx + r11*tyy + r12*tzz + c_offset;
        Dtype bz = r20*txx + r21*tyy + r22*tzz + c_offset;

        if(bx >= 0 && bx < grid_dim
            && by >= 0 && by < grid_dim
            && bz >= 0 && bz < grid_dim) {
          int x0, y0, z0, x1, y1, z1;
          SnapGrid_gpu(bx, x0, x1, grid_dim_1);
          SnapGrid_gpu(by, y0, y1, grid_dim_1);
          SnapGrid_gpu(bz, z0, z1, grid_dim_1);
          Dtype x_x0 = bx-x0;
          Dtype y_y0 = by-y0;
          Dtype z_z0 = bz-z0;
          Dtype x1_x = x1-bx;
          Dtype y1_y = y1-by;
          Dtype z1_z = z1-bz;
          top_data[t_batch_idx*num_grids+t_grid_idx] = Interpolate_gpu(bottom_data, b_batch_idx, x0, y0, z0, x1, y1, z1, x_x0, y_y0, z_z0, x1_x, y1_y, z1_z, grid_dim, grid_dim, grid_dim);
        } else {
          top_data[t_batch_idx*num_grids+t_grid_idx] = pad_value;
        }
      } /* rotation_idx */
    } /* b_batch_idx */
  }
}

template <typename Dtype>
void RotateLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  ForwardLabel(bottom[1], top[1]);

  const Dtype* bottom_data = bottom[0]->gpu_data();
  const vector<int>& bottom_shape = bottom[0]->shape();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int batch_size = bottom_shape[0];
  const int grid_dim = bottom_shape[1];
  const int num_grids = bottom[0]->count(1);
  const Dtype* rotations_data = rotations_.gpu_data();

  // NOLINT_NEXT_LINE(whitespace/operators)
  RotateForward<Dtype><<<CAFFE_GET_BLOCKS(num_grids), CAFFE_CUDA_NUM_THREADS>>>(num_grids, grid_dim, batch_size, num_rotation_,
      pad_value_, bottom_data, rotations_data, top_data);
  CUDA_POST_KERNEL_CHECK;

  //Dtype amax, aavg;
  //caffe_gpu_amax(top[0]->count(), top[0]->gpu_data(), &amax);
  //caffe_gpu_aavg(top[0]->count(), top[0]->gpu_data(), &aavg);
  //LOG(INFO) << "RotateLayer::Forward_gpu top_data max-avg: " << amax << "\t" << aavg;
}

INSTANTIATE_LAYER_GPU_FUNCS(RotateLayer);


}  // namespace caffe
